#include <stdio.h>
#include <stdlib.h>
#include <assert.h>
#include <hip/hip_runtime.h>
#include <hipsolver.h>

#include "common.h"

void check_pivot(bool pivot_on){
    if (pivot_on){
        printf("pivot is on : compute P*A = L*U \n");
    }else{
        printf("pivot is off: compute A = L*U (not numerically stable)\n");
    }
}

void printMatrix(int m, int n, const double*A, int lda, const char* name)
{
    for(int row = 0 ; row < m ; row++){
        for(int col = 0 ; col < n ; col++){
            double Areg = A[row + col*lda];
            printf("%s(%d,%d) = %f\n", name, row+1, col+1, Areg);
        }
    }
}

double * generate_rhs(const int size){
    double * m = (double*) malloc(sizeof(double)*size);
    return m;
}


double * generate_squared_matrix(const int size){
    double * m = (double*) malloc(sizeof(double)*size*size);
    return m;
}

int main(int argc, char*argv[])
{
    hipsolverHandle_t cusolverH = NULL;
    hipStream_t stream = NULL;

    hipsolverStatus_t status = HIPSOLVER_STATUS_SUCCESS;
    hipError_t cudaStat1 = hipSuccess;
    hipError_t cudaStat2 = hipSuccess;
    hipError_t cudaStat3 = hipSuccess;
    hipError_t cudaStat4 = hipSuccess;
    const int m = 3;
    const int lda = m;
    const int ldb = m;
    const int size = lda*m;
    double *B = generate_rhs(size);
    double X[m]; /* X = A\B */
    double LU[lda*m]; /* L and U */
    int Ipiv[m];      /* host copy of pivoting sequence */
    int info = 0;     /* host copy of error info */

    double *d_A = NULL; /* device copy of A */
    double *d_B = NULL; /* device copy of B */
    int *d_Ipiv = NULL; /* pivoting sequence */
    int *d_info = NULL; /* error info */
    int  lwork = 0;     /* size of workspace */
    double *d_work = NULL; /* device workspace for getrf */

    double *A;
    int matrix_dim = 0;
    const char *input_file = NULL;
    handle_arguments(argc, argv, &matrix_dim, input_file);
    //Creating matrix A
    func_ret_t ret;
    if (input_file) {
        printf("Reading matrix from file %s\n", input_file);
        ret = create_matrix_from_file(&A, input_file, &matrix_dim);
        if (ret != RET_SUCCESS) {
            A = NULL;
            fprintf(stderr, "error create matrix from file %s\n", input_file);
            exit(EXIT_FAILURE);
        }
    }
    else if (matrix_dim) {
        printf("Creating matrix internally size=%d\n", matrix_dim);
        ret = create_matrix(&A, matrix_dim);
        if (ret != RET_SUCCESS) {
            A = NULL;
            fprintf(stderr, "error create matrix internally size=%d\n", matrix_dim);
            exit(EXIT_FAILURE);
        }
    }
    else {
        printf("No input file specified!\n");
        exit(EXIT_FAILURE);
    }

    const bool pivot_on = true;
    check_pivot(pivot_on);


    printf("A = (matlab base-1)\n");
    printMatrix(m, m, A, lda, "A");
    printf("=====\n");

    printf("B = (matlab base-1)\n");
    printMatrix(m, 1, B, ldb, "B");
    printf("=====\n");

    /* step 1: create cusolver handle, bind a stream */
    status = hipsolverDnCreate(&cusolverH);
    assert(HIPSOLVER_STATUS_SUCCESS == status);

    cudaStat1 = hipStreamCreateWithFlags(&stream, hipStreamNonBlocking);
    assert(hipSuccess == cudaStat1);

    status = hipsolverSetStream(cusolverH, stream);
    assert(HIPSOLVER_STATUS_SUCCESS == status);

    /* step 2: copy A to device */
    cudaStat1 = hipMalloc ((void**)&d_A, sizeof(double) * size);
    cudaStat2 = hipMalloc ((void**)&d_B, sizeof(double) * m);
    cudaStat4 = hipMalloc ((void**)&d_info, sizeof(int));
    assert(hipSuccess == cudaStat1);
    assert(hipSuccess == cudaStat2);
    assert(hipSuccess == cudaStat3);
    assert(hipSuccess == cudaStat4);

    cudaStat1 = hipMemcpy(d_A, A, sizeof(double)*size, hipMemcpyHostToDevice);
    cudaStat2 = hipMemcpy(d_B, B, sizeof(double)*m, hipMemcpyHostToDevice);
    assert(hipSuccess == cudaStat1);
    assert(hipSuccess == cudaStat2);

    /* step 3: query working space of getrf */
    status = hipsolverDnDgetrf_bufferSize(
            cusolverH,
            m,
            m,
            d_A,
            lda,
            &lwork);
    assert(HIPSOLVER_STATUS_SUCCESS == status);

    cudaStat1 = hipMalloc((void**)&d_work, sizeof(double)*lwork);
    assert(hipSuccess == cudaStat1);

    /* step 4: LU factorization */
    if (!pivot_on){
        d_Ipiv = NULL;
    }else{
        cudaStat2 = hipMalloc ((void**)&d_Ipiv, sizeof(int) * m);
        assert(hipSuccess == cudaStat2);
    }

    status = hipsolverDnDgetrf(
            cusolverH,
            m,
            m,
            d_A,
            lda,
            d_work,
            d_Ipiv,
            d_info);
    cudaStat1 = hipDeviceSynchronize();
    assert(HIPSOLVER_STATUS_SUCCESS == status);
    assert(hipSuccess == cudaStat1);

    if (pivot_on)
        cudaStat1 = hipMemcpy(Ipiv , d_Ipiv, sizeof(int)*m,
                hipMemcpyDeviceToHost);

    cudaStat2 = hipMemcpy(LU, d_A, sizeof(double)*lda*m,
            hipMemcpyDeviceToHost);
    cudaStat3 = hipMemcpy(&info, d_info, sizeof(int), hipMemcpyDeviceToHost);
    assert(hipSuccess == cudaStat1);
    assert(hipSuccess == cudaStat2);
    assert(hipSuccess == cudaStat3);

    if ( 0 > info ){
        printf("%d-th parameter is wrong \n", -info);
        exit(1);
    }
    if (pivot_on){
        printf("pivoting sequence, matlab base-1\n");
        for(int j = 0 ; j < m ; j++){
            printf("Ipiv(%d) = %d\n", j+1, Ipiv[j]);
        }
    }
    printf("L and U = (matlab base-1)\n");
    printMatrix(m, m, LU, lda, "LU");
    printf("=====\n");

    status = hipsolverDnDgetrs(
            cusolverH,
            HIPBLAS_OP_N,
            m,
            1, /* nrhs */
            d_A,
            lda,
            d_Ipiv,
            d_B,
            ldb,
            d_info);

    cudaStat1 = hipDeviceSynchronize();
    assert(HIPSOLVER_STATUS_SUCCESS == status);
    assert(hipSuccess == cudaStat1);

    cudaStat1 = hipMemcpy(X , d_B, sizeof(double)*m, hipMemcpyDeviceToHost);
    assert(hipSuccess == cudaStat1);

    printf("X = (matlab base-1)\n");
    printMatrix(m, 1, X, ldb, "X");
    printf("=====\n");

    /* free resources */
    if (d_A    ) hipFree(d_A);
    if (d_B    ) hipFree(d_B);
    if (d_Ipiv ) hipFree(d_Ipiv);
    if (d_info ) hipFree(d_info);
    if (d_work ) hipFree(d_work);

    if (cusolverH   ) hipsolverDnDestroy(cusolverH);
    if (stream      ) hipStreamDestroy(stream);

    hipDeviceReset();

    return 0;
}

