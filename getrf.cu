#include <stdio.h>
#include <stdlib.h>
#include <assert.h>
#include <hip/hip_runtime.h>
#include <hipsolver.h>

#include "hip/hip_runtime_api.h"
#include "common.h"

void check_pivot(bool pivot_on){
    if (pivot_on){
        printf("pivot is on : compute P*A = L*U \n");
    }else{
        printf("pivot is off: compute A = L*U (not numerically stable)\n");
    }
}

void printMatrix(int m, int n, const double*A, int lda, const char* name)
{
    for(int row = 0 ; row < m ; row++){
        for(int col = 0 ; col < n ; col++){
            double Areg = A[row + col*lda];
            printf("%s(%d,%d) = %f\n", name, row+1, col+1, Areg);
        }
    }
}

int main(int argc, char*argv[])
{
    hipsolverHandle_t cusolverH = NULL;
    hipStream_t stream = NULL;

    hipsolverStatus_t status = HIPSOLVER_STATUS_SUCCESS;
    hipError_t cudaStat1 = hipSuccess;
    hipError_t cudaStat2 = hipSuccess;
    hipError_t cudaStat3 = hipSuccess;
    hipError_t cudaStat4 = hipSuccess;
    int info = 0;     /* host copy of error info */

    double *d_A = NULL; /* device copy of A */
    double *d_B = NULL; /* device copy of B */
    int *d_Ipiv = NULL; /* pivoting sequence */
    int *d_info = NULL; /* error info */
    int  lwork = 0;     /* size of workspace */
    double *d_work = NULL; /* device workspace for getrf */

    double *A,*B;
    int matrix_dim = 0;
    int rhs_matrix_dim = 0;
    const char *input_file = NULL;
    const char *rhs_input_file = NULL;
    handle_arguments(argc, argv, &matrix_dim, input_file, &rhs_matrix_dim, rhs_input_file);
    //Creating matrix A
    func_ret_t ret;
    if (input_file) {
        printf("Reading matrix from file %s\n", input_file);
        ret = create_matrix_from_file(&A, input_file, &matrix_dim);
        if (ret != RET_SUCCESS) {
            A = NULL;
            fprintf(stderr, "error create matrix from file %s\n", input_file);
            exit(EXIT_FAILURE);
        }
    }
    else if (matrix_dim) {
        printf("Creating matrix internally size=%d\n", matrix_dim);
        ret = create_matrix_from_random(&A, matrix_dim);
        if (ret != RET_SUCCESS) {
            A = NULL;
            fprintf(stderr, "error create matrix internally size=%d\n", matrix_dim);
            exit(EXIT_FAILURE);
        }
    }
    else {
        printf("No input file specified!\n");
        exit(EXIT_FAILURE);
    }

    if (rhs_input_file) {
        ret = create_rhs_matrix_from_file(&B, rhs_input_file, &rhs_matrix_dim);
        if (ret != RET_SUCCESS) {
            B = NULL;
            fprintf(stderr, "error create matrix from file %s\n", rhs_input_file);
            exit(EXIT_FAILURE);
        }
    }
    else if (rhs_matrix_dim) {
        printf("Creating RHS matrix internally size=%d\n", rhs_matrix_dim);
        ret = create_rhs_matrix_from_random(&B, rhs_matrix_dim);
        if (ret != RET_SUCCESS) {
            B = NULL;
            fprintf(stderr, "error create matrix internally size=%d\n", rhs_matrix_dim);
            exit(EXIT_FAILURE);
        }
    }
    else {
        printf("No input file specified!\n");
        exit(EXIT_FAILURE);
    }

    const int size = matrix_dim*matrix_dim;
    const int lda = matrix_dim;
    const int ldb = matrix_dim;
    double X[matrix_dim]; /* X = A\B */
    double LU[lda*matrix_dim]; /* L and U */
    int Ipiv[matrix_dim];      /* host copy of pivoting sequence */

    const bool pivot_on = true;
    float getrf_msecTotal = 0.0f, getrs_msecTotal = 0.0f;
    check_pivot(pivot_on);


    printf("A = (matlab base-1)\n");
    printMatrix(matrix_dim, matrix_dim, A, lda, "A");
    printf("=====\n");

    printf("B = (matlab base-1)\n");
    printMatrix(matrix_dim, 1, B, ldb, "B");
    printf("=====\n");

    /* step 1: create cusolver handle, bind a stream */
    status = hipsolverDnCreate(&cusolverH);
    assert(HIPSOLVER_STATUS_SUCCESS == status);

    cudaStat1 = hipStreamCreateWithFlags(&stream, hipStreamNonBlocking);
    assert(hipSuccess == cudaStat1);

    status = hipsolverSetStream(cusolverH, stream);
    assert(HIPSOLVER_STATUS_SUCCESS == status);

    /* step 2: copy A to device */
    cudaStat1 = hipMalloc ((void**)&d_A, sizeof(double) * matrix_dim *matrix_dim );
    cudaStat2 = hipMalloc ((void**)&d_B, sizeof(double) * matrix_dim);
    cudaStat4 = hipMalloc ((void**)&d_info, sizeof(int));
    assert(hipSuccess == cudaStat1);
    assert(hipSuccess == cudaStat2);
    assert(hipSuccess == cudaStat3);
    assert(hipSuccess == cudaStat4);

    cudaStat1 = hipMemcpy(d_A, A, sizeof(double)*size, hipMemcpyHostToDevice);
    cudaStat2 = hipMemcpy(d_B, B, sizeof(double)*matrix_dim, hipMemcpyHostToDevice);
    assert(hipSuccess == cudaStat1);
    assert(hipSuccess == cudaStat2);

    /* step 3: query working space of getrf */
    status = hipsolverDnDgetrf_bufferSize(
            cusolverH,
            matrix_dim,
            matrix_dim,
            d_A,
            lda,
            &lwork);
    assert(HIPSOLVER_STATUS_SUCCESS == status);

    cudaStat1 = hipMalloc((void**)&d_work, sizeof(double)*lwork);
    assert(hipSuccess == cudaStat1);

    /* step 4: LU factorization */
    if (!pivot_on){
        d_Ipiv = NULL;
    }else{
        cudaStat2 = hipMalloc ((void**)&d_Ipiv, sizeof(int) * matrix_dim);
        assert(hipSuccess == cudaStat2);
    }

    hipblasHandle_t handle;
    // Allocate CUDA events that we'll use for timing
    hipblasCreate(&handle);
    hipEvent_t start;
    checkCudaErrors(hipEventCreate(&start));
    hipEvent_t stop;
    checkCudaErrors(hipEventCreate(&stop));

    hipDeviceSynchronize();
    // Record the start event
    checkCudaErrors(hipEventRecord(start, NULL));
    status = hipsolverDnDgetrf(
            cusolverH,
            matrix_dim,
            matrix_dim,
            d_A,
            lda,
            d_work,
            d_Ipiv,
            d_info);
    cudaStat1 = hipDeviceSynchronize();
    // Record the stop event
    checkCudaErrors(hipEventRecord(stop, NULL));
    // Wait for the stop event to complete
    checkCudaErrors(hipEventSynchronize(stop));
    checkCudaErrors(hipEventElapsedTime(&getrf_msecTotal, start, stop));

    assert(HIPSOLVER_STATUS_SUCCESS == status);
    assert(hipSuccess == cudaStat1);

    if (pivot_on)
        cudaStat1 = hipMemcpy(Ipiv , d_Ipiv, sizeof(int)*matrix_dim,
                hipMemcpyDeviceToHost);

    cudaStat2 = hipMemcpy(LU, d_A, sizeof(double)*lda*matrix_dim,
            hipMemcpyDeviceToHost);
    cudaStat3 = hipMemcpy(&info, d_info, sizeof(int), hipMemcpyDeviceToHost);
    assert(hipSuccess == cudaStat1);
    assert(hipSuccess == cudaStat2);
    assert(hipSuccess == cudaStat3);

    if ( 0 > info ){
        printf("%d-th parameter is wrong \n", -info);
        exit(1);
    }
    if (pivot_on){
        printf("pivoting sequence, matlab base-1\n");
        for(int j = 0 ; j < matrix_dim ; j++){
            printf("Ipiv(%d) = %d\n", j+1, Ipiv[j]);
        }
    }
    printf("L and U = (matlab base-1)\n");
    printMatrix(matrix_dim, matrix_dim, LU, lda, "LU");
    printf("=====\n");

    hipDeviceSynchronize();
    checkCudaErrors(hipEventRecord(start, NULL));
    status = hipsolverDnDgetrs(
            cusolverH,
            HIPBLAS_OP_N,
            matrix_dim,
            1, /* nrhs */
            d_A,
            lda,
            d_Ipiv,
            d_B,
            ldb,
            d_info);

    cudaStat1 = hipDeviceSynchronize();
    checkCudaErrors(hipEventRecord(stop, NULL));
    checkCudaErrors(hipEventSynchronize(stop));
    checkCudaErrors(hipEventElapsedTime(&getrs_msecTotal, start, stop));
    assert(HIPSOLVER_STATUS_SUCCESS == status);
    assert(hipSuccess == cudaStat1);

    cudaStat1 = hipMemcpy(X , d_B, sizeof(double)*matrix_dim, hipMemcpyDeviceToHost);
    assert(hipSuccess == cudaStat1);

    printf("X = (matlab base-1)\n");
    printMatrix(matrix_dim, 1, X, ldb, "X");
    printf("=====\n");

    /* free resources */
    if (d_A    ) hipFree(d_A);
    if (d_B    ) hipFree(d_B);
    if (d_Ipiv ) hipFree(d_Ipiv);
    if (d_info ) hipFree(d_info);
    if (d_work ) hipFree(d_work);

    if (cusolverH   ) hipsolverDnDestroy(cusolverH);
    if (stream      ) hipStreamDestroy(stream);

    hipblasDestroy(handle);
    hipDeviceReset();

    return 0;
}

