#include "hip/hip_runtime.h"
#include <cstdio>
#include <hiprand.h>
#include <hip/hip_runtime.h>
#include <cassert>
#include "utils.hpp"

/* Generate positive eignenvalues, singular values are arithmetically distributed
 * Sigma_i = 1 - (\frac{i - 1}{n-1})(1-\frac{1}{cond})
 * where cond is the condition number of the matrix with these eigenvalues.
 * Condition number is defined with the infinite norm as: cond = norm(A) \times norm(A^{-1})
 *
 * Matrix D heap memory management must be handle outside the function.
 * D is a square matrix of size n by n.
 */
__global__
void
fill_diagonal_matrix(DATATYPE * d_D, int n, DATATYPE cond){
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if(idx%(n+1) == 0)
        d_D[idx] = SIGMA_I(idx%n,n,cond);
    else
        d_D[idx] = 0;
}

void generate_diagonal_matrix(DATATYPE ** d_D, int n, DATATYPE cond){
    hipError_t cudaStat1 = hipSuccess;
    cudaStat1 = hipMalloc((void**)&(*d_D), sizeof(DATATYPE)*n*n);
    assert(hipSuccess == cudaStat1);
    // Fill D with diagonal values
    fill_diagonal_matrix<<<1,n*n>>>(*d_D, n, cond);
}


/* Generate matrix */
void fill_random_matrix(double *A, int N){
    hiprandGenerator_t prng;
    hiprandCreateGenerator (&prng, HIPRAND_RNG_PSEUDO_DEFAULT);
    hiprandSetPseudoRandomGeneratorSeed(prng, (unsigned long long) clock());
    hiprandGenerateUniformDouble(prng, A, N);
}

void orthogonalize_matrix(DATATYPE * d_A, int n){
}

/* Matrix Q heap memory management must be handle outside the function.
 * Q is a square matrix of size n by n.
 */
void generate_orthogonal_matrix(DATATYPE ** d_Q, int n, DATATYPE cond){
    hipError_t cudaStat1 = hipSuccess;
    cudaStat1 = hipMalloc((void**)&(*d_Q), sizeof(DATATYPE)*n*n);
    assert(hipSuccess == cudaStat1);
    // Reset to 0 values inside Q
    fill_random_matrix(*d_Q,n);
    orthogonalize_matrix(*d_Q,n);
}

void ConstantInitDouble(double *data, int size, double val) {
    for (int i = 0; i < size; ++i) {
        data[i] = val+i;
    }
}
